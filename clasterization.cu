#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<iostream>
#include <SFML/Graphics.hpp>

#define K 10
#define IC 1000
#define BLOCK_SIZE 16
#define NUM_THREADS 100

struct Point
{
    //coordinates
    int x;
    int y;
    //color
    int r;
    int g;
    int b;
    //centroids number
    int number;
};
struct Centroid
{
    int r;
    int g;
    int b;
};

struct Matrix
{
    int x;
    int y;
    Centroid * elements;
};


Centroid* initializeCentroids(Matrix image)
{
    Centroid* centroids = (Centroid*)malloc(sizeof(Centroid)*K);
    for (int i = 0; i < K; i++)
        centroids[i] = image.elements[(rand() % image.x)*image.y + (rand() % image.y)];
    return centroids;
};


__global__ void findClosestCentroids_cuda(Matrix image, Centroid* centroids, Point* res,int start_x = 0 ,int start_y = 0)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x + start_x;
    int row =  blockIdx.y * blockDim.y + threadIdx.y + start_y;

    int n = col* image.y + row; //текущий номер пикселя
    res[n].x = col;
    res[n].y = row;
    res[n].r = image.elements[n].r;
    res[n].g = image.elements[n].g;
    res[n].b = image.elements[n].b;
    int min_dist = 200000000;
    for (int k = 0; k < K; k++)
    {
        int distance_2 = (res[n].r - centroids[k].r) * (res[n].r - centroids[k].r)
            + (res[n].g - centroids[k].g) * (res[n].g - centroids[k].g)
            + (res[n].b - centroids[k].b) * (res[n].b - centroids[k].b);
        if (distance_2 < min_dist)
        {
            min_dist = distance_2;
            res[n].number = k;
        }
    }
}

__global__ void ComputeMeans_cuda(const Point* ids, int imsize, Centroid* res)
{
    __shared__ int num[K*NUM_THREADS];
    __shared__ int r[K*NUM_THREADS];
    __shared__ int g[K*NUM_THREADS];
    __shared__ int b[K*NUM_THREADS];
    int id = threadIdx.x;
    for (int i = 0; i < K; i++)
    {
        r[i+id*K] = 0;
        g[i+id*K] = 0;
        b[i+id*K] = 0;
        num[i+id*K] = 0;
    }
    int block_size = imsize/NUM_THREADS;

    for (int i = block_size*id; i < block_size*(id+1); i++)
    {
        r[ids[i].number + id*K] += (int)ids[i].r;
        g[ids[i].number + id*K] += (int)ids[i].g;
        b[ids[i].number + id*K] += (int)ids[i].b;
        num[ids[i].number + id*K]++;
    }
    if (0 == id)
    {
        for (int i = block_size*NUM_THREADS; i < imsize; i++)
        {
            r[ids[i].number + id*K] += (int)ids[i].r;
            g[ids[i].number + id*K] += (int)ids[i].g;
            b[ids[i].number + id*K] += (int)ids[i].b;
            num[ids[i].number + id*K]++;
        }
    }
    __syncthreads();
    if (0 == id)
    {
        for (int i = 1; i < NUM_THREADS; i++)
        {
            for (int j = 0; j < K; j++)
            {
                r[j] += r[j+i*K];
                g[j] += g[j+i*K];
                b[j] += b[j+i*K];
                num[j] += num[j+i*K];
            }
        }
        for (int i = 0; i < K; i++)
        {
            r[i] /= num[i];
            g[i] /= num[i];
            b[i] /= num[i];
            res[i].r = r[i];
            res[i].g = g[i];
            res[i].b = b[i];
        }
    }

}

__global__ void ChangeColors_cuda(Matrix image, Point*  ids, Centroid* centroids,int start_x =0, int start_y = 0)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x + start_x;
    int row =  blockIdx.y * blockDim.y + threadIdx.y + start_y;
    int n = col* image.y + row; //текущий номер пикселя
    image.elements[ids[n].x* image.y + ids[n].y] = centroids[ids[n].number];
}

int main(void)
{
    //loading picture using sfml
    sf::Image im;
    sf::Texture texture;
    sf::Sprite photo;
    texture.loadFromFile("/home/cuda/file.png");
    sf::Image image_sf = texture.copyToImage();
    // prepare host data
    int x = image_sf.getSize().x;
    int y = image_sf.getSize().y;
    Matrix image;
    image.x =x;
    image.y= y;
    image.elements = (Centroid*)malloc(sizeof(Centroid)*x*y);

    for (int i = 0 ; i < x ; i ++)
    {
        for (int j = 0 ; j < y; j++)
        {
            sf::Color pix = image_sf.getPixel(i,j);
            image.elements[i* image.y + j].r = pix.r;
            image.elements[i* image.y + j].g = pix.g;
            image.elements[i* image.y + j].b = pix.b;
        }
    }

    Centroid* centroids = initializeCentroids(image);

    // prepare device data
    Matrix image_cuda;
    image_cuda.x = image.x;
    image_cuda.y = image.y;

    Centroid* centroids_cuda;

    hipMalloc(&centroids_cuda,sizeof(Centroid)*K);
    hipMalloc(&image_cuda.elements,sizeof(Centroid)*x*y);

    hipMemcpy(centroids_cuda,centroids, sizeof(Centroid)*K,hipMemcpyHostToDevice);
    hipMemcpy(image_cuda.elements,image.elements,sizeof(Centroid)*x*y,hipMemcpyHostToDevice);

    Point* ids_cuda;
    hipMalloc(&ids_cuda,image.x*image.y*sizeof(Point));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(image_cuda.x / dimBlock.x, image_cuda.y / dimBlock.y);

    dim3 dimBlockE1(image_cuda.x % dimBlock.x, image_cuda.y % dimBlock.y);
    dim3 dimGridE1(1,1);
    if (0 != dimBlockE1.y)
        dimGridE1 = dim3(1, image_cuda.y/dimBlockE1.y);

    dim3 dimBlockE2(image_cuda.x % dimBlock.x, image_cuda.y % dimBlock.y);
    dim3 dimGridE2(1,1);
    if (0 != dimBlockE2.x)
        dimGridE2 = dim3(image_cuda.x/dimBlockE2.x, 1);
    // process
    for (int it = 0; it < IC; it++)
    {
        findClosestCentroids_cuda<<<dimGrid, dimBlock>>>(image_cuda, centroids_cuda, ids_cuda);
        //Edges we process separately
        if (0 != dimBlockE1.y)
        {
            findClosestCentroids_cuda <<< dimGridE1, dimBlockE1 >>> (image_cuda, centroids_cuda, ids_cuda,
                image_cuda.x - image_cuda.x % dimBlock.x,
                0);
        }
        if (0 != dimBlockE2.x)
        {
            findClosestCentroids_cuda <<< dimGridE2, dimBlockE2 >>> (image_cuda, centroids_cuda, ids_cuda,
                0,
                image_cuda.y - image_cuda.y % dimBlock.y);
        }

        ComputeMeans_cuda<<<1,NUM_THREADS>>>(ids_cuda,x*y,centroids_cuda);

    }
    ChangeColors_cuda<<<dimGrid, dimBlock>>>(image_cuda,ids_cuda,centroids_cuda);
    //Edges we process separately
    if (0 != dimBlockE1.y)
    {
        ChangeColors_cuda << < dimGridE1, dimBlockE1 >> > (image_cuda, ids_cuda, centroids_cuda,
            image_cuda.x - image_cuda.x % dimBlock.x,
            0);
    }
    if (0 != dimBlockE2.x)
    {
        ChangeColors_cuda <<< dimGridE2, dimBlockE2 >>> (image_cuda, ids_cuda, centroids_cuda,
            0,
            image_cuda.y - image_cuda.y % dimBlock.y);
    }

    hipMemcpy(image.elements,image_cuda.elements,sizeof(Centroid)*x*y,hipMemcpyDeviceToHost);
    hipFree(centroids_cuda);
    hipFree(image_cuda.elements);
    hipFree(ids_cuda);

    //return data to sfml format
    for (int i = 0 ; i < x ; i ++)
    {
        for (int j = 0 ; j < y; j++)
        {
            image_sf.setPixel(i,j,sf::Color( image.elements[i* image.y + j].r,
                                             image.elements[i* image.y + j].g,
                                             image.elements[i* image.y + j].b));
        }
    }

    image_sf.saveToFile("/home/cuda/result.png");
    texture.update(image_sf);
    photo.setTexture(texture);
}



